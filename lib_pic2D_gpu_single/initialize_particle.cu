#include "hip/hip_runtime.h"
#include "initialize_particle.hpp"
#include <thrust/transform.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <random>


__global__ void uniformForPositionX_kernel(
    Particle* particle, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, i, 0, &state);
        float x = hiprand_uniform(&state) * (device_xmax - device_xmin) + device_xmin;
        particle[i + nStart].x = x;
    }
}

void InitializeParticle::uniformForPositionX(
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    uniformForPositionX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}


__global__ void uniformForPositionY_kernel(
    Particle* particle, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, i, 0, &state);
        float y = hiprand_uniform(&state) * (device_ymax - device_ymin) + device_ymin;
        particle[i + nStart].y = y;
    }
}

void InitializeParticle::uniformForPositionY(
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    uniformForPositionY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}


__global__ void maxwellDistributionForVelocity_kernel(
    Particle* particle, 
    const float bulkVxSpecies, const float bulkVySpecies, const float bulkVzSpecies, 
    const float vxThSpecies, const float vyThSpecies, const float vzThSpecies, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz; 
        hiprand_init(seed,           100 * i, 0, &stateVx);
        hiprand_init(seed + 1000000, 100 * i, 0, &stateVy);
        hiprand_init(seed + 2000000, 100 * i, 0, &stateVz);

        float vx, vy, vz;

        while (true) {
            vx = bulkVxSpecies + hiprand_normal(&stateVx) * vxThSpecies;
            vy = bulkVySpecies + hiprand_normal(&stateVy) * vyThSpecies;
            vz = bulkVzSpecies + hiprand_normal(&stateVz) * vzThSpecies;

            if (vx * vx + vy * vy + vz * vz < device_c * device_c) break;
        }

        particle[i + nStart].vx = vx;
        particle[i + nStart].vy = vy;
        particle[i + nStart].vz = vz;
        particle[i + nStart].gamma = sqrt(1.0f + (vx * vx + vy * vy + vz * vz) / (device_c * device_c));
    }
}


void InitializeParticle::maxwellDistributionForVelocity(
    float bulkVxSpecies, 
    float bulkVySpecies, 
    float bulkVzSpecies, 
    float vxThSpecies, 
    float vyThSpecies, 
    float vzThSpecies, 
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    maxwellDistributionForVelocity_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        bulkVxSpecies, bulkVySpecies, bulkVzSpecies, 
        vxThSpecies, vyThSpecies, vzThSpecies, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}


__global__ void harrisForPositionY_kernel(
    Particle* particle, float sheatThickness, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, 10 * i, 0, &state);
        float yCenter = 0.5f * (device_ymax - device_ymin) + device_ymin;

        float randomValue;
        float y;
        while (true) {
            randomValue = hiprand_uniform(&state);
            y = yCenter + sheatThickness * atanh(2.0f * randomValue - 1.0f);

            if (device_ymin < y && y < device_ymax) break;
        }
        
        particle[i + nStart].y = y;
    }
}

void InitializeParticle::harrisForPositionY(
    int nStart, 
    int nEnd, 
    int seed, 
    float sheatThickness, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    harrisForPositionY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), sheatThickness, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}


__global__ void harrisBackgroundForPositionY_kernel(
    Particle* particle, float sheatThickness, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, 10 * i, 0, &state);
        float yCenter = 0.5f * (device_ymax - device_ymin) + device_ymin;

        float randomValue;
        float y;
        while (true) {
            randomValue = hiprand_uniform(&state);
            y = randomValue * (device_ymax - device_ymin);

            if (randomValue < (1.0f - 1.0f / cosh((y - yCenter) / sheatThickness))) break;
        } 
        
        particle[i + nStart].y = y;
    }
}

void InitializeParticle::harrisBackgroundForPositionY(
    int nStart, 
    int nEnd, 
    int seed, 
    float sheatThickness, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    harrisBackgroundForPositionY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), sheatThickness, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}

