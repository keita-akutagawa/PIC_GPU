#include "hip/hip_runtime.h"
#include "initialize_particle.hpp"
#include <thrust/transform.h>
#include <thrust/random.h>
#include <cmath>


__global__ void uniformForPositionX_kernel(
    Particle* particle, 
    const int nStart, const int nEnd, const int seed
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        thrust::default_random_engine rng(i + nStart);
        thrust::uniform_real_distribution<float> dist(device_xmin + 1e-20f, device_xmax - 1e-20f);
        particle[i + nStart].x = dist(rng);
    }
}


void InitializeParticle::uniformForPositionX(
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    uniformForPositionX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}



__global__ void maxwellDistributionForVelocity_kernel(
    Particle* particle, 
    const float bulkVxSpecies, const float bulkVySpecies, const float bulkVzSpecies, const float vThSpecies, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        thrust::default_random_engine rng(nStart + i);
        thrust::random::normal_distribution<float> dist_vx(bulkVxSpecies, vThSpecies);
        thrust::random::normal_distribution<float> dist_vy(bulkVySpecies, vThSpecies);
        thrust::random::normal_distribution<float> dist_vz(bulkVzSpecies, vThSpecies);

        float vx, vy, vz;

        while (true) {
            vx = dist_vx(rng);
            vy = dist_vy(rng);
            vz = dist_vz(rng);

            if (vx * vx + vy * vy + vz * vz < device_c * device_c) break;
        }

        particle[i + nStart].vx = vx;
        particle[i + nStart].vy = vy;
        particle[i + nStart].vz = vz;
        particle[i + nStart].gamma = sqrt(1.0 + (vx * vx + vy * vy + vz * vz) / (device_c * device_c));
    }
}


void InitializeParticle::maxwellDistributionForVelocity(
    float bulkVxSpecies, 
    float bulkVySpecies, 
    float bulkVzSpecies, 
    float vThSpecies, 
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    maxwellDistributionForVelocity_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        bulkVxSpecies, bulkVySpecies, bulkVzSpecies, vThSpecies, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}

