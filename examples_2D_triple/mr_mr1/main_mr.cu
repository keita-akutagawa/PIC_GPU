#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <cmath>
#include "../../lib_pic2D_gpu_single_triple/pic2D.hpp"
#include <hip/hip_runtime.h>


std::string directoryname = "results_mr_triple_4-16-20";
std::string filenameWithoutStep = "mr";
std::ofstream logfile("results_mr_triple_4-16-20/log_mr.txt");

const int totalStep = 60000;
const int fieldRecordStep = 400;
const bool isParticleRecord = false;
const int particleRecordStep = totalStep;
float totalTime = 0.0f;

const float c = 1.0f;
const float epsilon0 = 1.0f;
const float mu0 = 1.0f;
const float dOfLangdonMarderTypeCorrection = 0.005f;

const int numberDensityIon = 50;
const int numberDensityHeavyIon = 2; //backgroundでの数密度を設定すること
const int numberDensityElectron = 50;

const float B0 = sqrt(static_cast<float>(numberDensityElectron)) / 1.0f;

const float mRatio = 1.0f;
const float mElectron = 1.0f;
const float mIon = mRatio * mElectron;
const float mHeavyIon = mIon * 100.0f;

const float tRatio = 1.0f;
const float tElectron = (B0 * B0 / 2.0 / mu0) / (numberDensityIon + numberDensityElectron * tRatio);
const float tIon = tRatio * tElectron;
const float tHeavyIon = tIon;

const float qRatio = -1.0f;
const float qElectron = -1.0f * sqrt(epsilon0 * tElectron / static_cast<float>(numberDensityElectron));
const float qIon = qRatio * qElectron;
const float qHeavyIon = qIon;

const float omegaPe = sqrt(static_cast<float>(numberDensityElectron) * pow(qElectron, 2) / mElectron / epsilon0);
const float omegaPi = sqrt(static_cast<float>(numberDensityIon) * pow(qIon, 2) / mIon / epsilon0);
const float omegaCe = abs(qElectron * B0 / mElectron);
const float omegaCi = qIon * B0 / mIon;

const float debyeLength = sqrt(epsilon0 * tElectron / static_cast<float>(numberDensityElectron) / pow(qElectron, 2));
//追加
const float ionInertialLength = c / omegaPi;

const int nx = int(1000.0f * ionInertialLength);
const float dx = 1.0f;
const float xmin = 0.0f * dx; 
const float xmax = nx * dx - 0.0f * dx;
//const float xmin = 0.5f * dx; 
//const float xmax = nx * dx - 1.5f * dx;

const int ny = int(500.0f * ionInertialLength);
const float dy = 1.0f;
const float ymin = 1.0f * dy; 
const float ymax = ny * dy - 1.5f * dy;

const float dt = 0.25f;

//追加
const float sheatThickness = 2.0f * ionInertialLength;
const float triggerRatio = 0.0f;
const float xPointPosition = 250.0f * ionInertialLength;

//追加
const int harrisNumIon = int(nx * numberDensityIon * 2.0f * sheatThickness);
const int backgroundNumIon = int(0.16f * nx * ny * numberDensityIon);
const long long totalNumIon = harrisNumIon + backgroundNumIon;
const int harrisNumElectron = int(nx * numberDensityElectron * 2.0f * sheatThickness);
const int backgroundNumElectron = int(0.2f * nx * ny * numberDensityElectron);
const long long totalNumElectron = harrisNumElectron + backgroundNumElectron;
const int backgroundNumHeavyIon = int(nx * ny * numberDensityHeavyIon);
const long long totalNumHeavyIon = backgroundNumHeavyIon;
const long long totalNumParticles = totalNumIon + totalNumElectron + totalNumHeavyIon;

const float vThIon = sqrt(2.0f * tIon / mIon);
const float vThElectron = sqrt(2.0f * tElectron / mElectron);
const float vThHeavyIon = sqrt(2.0f * tHeavyIon / mHeavyIon);

const float bulkVxElectron = 0.0f;
const float bulkVyElectron = 0.0f;
const float bulkVzElectron = c * debyeLength / sheatThickness * sqrt(2.0f / (1.0f + 1.0f/tRatio));
const float bulkVxIon = -bulkVxElectron / tRatio;
const float bulkVyIon = -bulkVyElectron / tRatio;
const float bulkVzIon = -bulkVzElectron / tRatio;

const float vThIonB = sqrt(2.0f * tIon * 0.2f / mIon);
const float vThElectronB = sqrt(2.0f * tElectron * 0.2f / mElectron);
const float vThHeavyIonB = sqrt(2.0f * tHeavyIon * 0.2f / mHeavyIon);

const float bulkVxElectronB = 0.0f;
const float bulkVyElectronB = 0.0f;
const float bulkVzElectronB = 0.0f;
const float bulkVxIonB = 0.0f;
const float bulkVyIonB = 0.0f;
const float bulkVzIonB = 0.0f;
const float bulkVxHeavyIonB = 0.0f;
const float bulkVyHeavyIonB = 0.0f;
const float bulkVzHeavyIonB = 0.0f;


__constant__ int device_totalStep;
__constant__ int device_recordStep;
__device__ float device_totalTime;

__constant__ float device_c;
__constant__ float device_epsilon0;
__constant__ float device_mu0;
__constant__ float device_dOfLangdonMarderTypeCorrection;

__constant__ int device_numberDensityIon;
__constant__ int device_numberDensityElectron;
__constant__ int device_numberDensityHeavyIon;

__constant__ float device_B0;

__constant__ float device_mRatio;
__constant__ float device_mElectron;
__constant__ float device_mIon;
__constant__ float device_mHeavyIon;

__constant__ float device_tRatio;
__constant__ float device_tElectron;
__constant__ float device_tIon;
__constant__ float device_tHeavyIon;

__constant__ float device_qRatio;
__constant__ float device_qElectron;
__constant__ float device_qIon;
__constant__ float device_qHeavyIon;

__constant__ float device_omegaPe;
__constant__ float device_omegaPi;
__constant__ float device_omegaCe;
__constant__ float device_omegaCi;

__constant__ float device_debyeLength;
//追加
__constant__ float device_ionInertialLength;

__constant__ int device_nx;
__constant__ float device_dx;
__constant__ float device_xmin;
__constant__ float device_xmax;

__constant__ int device_ny;
__constant__ float device_dy;
__constant__ float device_ymin;
__constant__ float device_ymax;

__constant__ float device_dt;

//追加
__constant__ float device_sheatThickness;
__constant__ float device_triggerRatio;
__constant__ float device_xPointPosition;

//追加
__constant__ int device_harrisNumIon;
__constant__ int device_backgroundNumIon;
__constant__ long long device_totalNumIon;
__constant__ int device_harrisNumElectron;
__constant__ int device_backgroundNumElectron;
__constant__ long long device_totalNumElectron;
__constant__ int device_backgroundNumHeavyIon;
__constant__ long long device_totalNumHeavyIon;
__constant__ long long device_totalNumParticles;

__constant__ float device_vThIon;
__constant__ float device_vThElectron;
__constant__ float device_vThHeavyIon;
__constant__ float device_bulkVxElectron;
__constant__ float device_bulkVyElectron;
__constant__ float device_bulkVzElectron;
__constant__ float device_bulkVxIon;
__constant__ float device_bulkVyIon;
__constant__ float device_bulkVzIon;
__constant__ float device_bulkVxHeavyIon;
__constant__ float device_bulkVyHeavyIon;
__constant__ float device_bulkVzHeavyIon;

__constant__ float device_vThIonB;
__constant__ float device_vThElectronB;
__constant__ float device_vThHeavyIonB;

__constant__ float device_bulkVxElectronB;
__constant__ float device_bulkVyElectronB;
__constant__ float device_bulkVzElectronB;
__constant__ float device_bulkVxIonB;
__constant__ float device_bulkVyIonB;
__constant__ float device_bulkVzIonB;
__constant__ float device_bulkVxHeavyIonB;
__constant__ float device_bulkVyHeavyIonB;
__constant__ float device_bulkVzHeavyIonB;


__global__ void initializeField_kernel(
    ElectricField* E, MagneticField* B
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx && j < device_ny) {
        float yCenter = 0.5f * (device_ymax - device_ymin) + device_ymin;
        E[j + device_ny * i].eX = 0.0f;
        E[j + device_ny * i].eY = 0.0f;
        E[j + device_ny * i].eZ = 0.0f;
        B[j + device_ny * i].bX = device_B0 * tanh((j * device_dy - yCenter) / device_sheatThickness)
                                - device_B0 * device_triggerRatio * (j * device_dy - yCenter) / device_sheatThickness
                                * exp(-(pow((i * device_dx - device_xPointPosition), 2) + pow((j * device_dy - yCenter), 2))
                                / pow(2.0f * device_sheatThickness, 2));
        B[j + device_ny * i].bY = device_B0 * device_triggerRatio * (i * device_dx - device_xPointPosition) / device_sheatThickness
                                * exp(-(pow((i * device_dx - device_xPointPosition), 2) + pow((j * device_dy - yCenter), 2))
                                / pow(2.0f * device_sheatThickness, 2)); 
        B[j + device_ny * i].bZ = 0.0f * device_B0;
    }
}

void PIC2D::initialize()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_ionInertialLength), &ionInertialLength, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_sheatThickness), &sheatThickness, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_triggerRatio), &triggerRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xPointPosition), &xPointPosition, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_harrisNumIon), &harrisNumIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_backgroundNumIon), &backgroundNumIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_harrisNumElectron), &harrisNumElectron, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_backgroundNumElectron), &backgroundNumElectron, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_backgroundNumHeavyIon), &backgroundNumHeavyIon, sizeof(int));

    initializeParticle.uniformForPositionX(
        0, totalNumIon, 0, particlesIon
    );
    initializeParticle.uniformForPositionX(
        0, totalNumElectron, 100, particlesElectron
    );
    initializeParticle.uniformForPositionX(
        0, totalNumHeavyIon, 200, particlesHeavyIon
    );

    initializeParticle.harrisForPositionY(
        0, harrisNumIon, 300, sheatThickness, particlesIon
    );
    initializeParticle.uniformForPositionY(
        harrisNumIon, totalNumIon, 400,particlesIon
    );
    initializeParticle.harrisForPositionY(
        0, harrisNumElectron, 500, sheatThickness, particlesElectron
    );
    initializeParticle.uniformForPositionY(
        harrisNumElectron, totalNumElectron, 600, particlesElectron
    );
    initializeParticle.uniformForPositionY(
        0, totalNumHeavyIon, 700, particlesHeavyIon
    );

    initializeParticle.maxwellDistributionForVelocity(
        bulkVxIon, bulkVyIon, bulkVzIon, vThIon, vThIon, vThIon, 
        0, harrisNumIon, 800, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxIonB, bulkVyIonB, bulkVzIonB, vThIonB, vThIonB, vThIonB, 
        harrisNumIon, totalNumIon, 900, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxElectron, bulkVyElectron, bulkVzElectron, vThElectron, vThElectron, vThElectron, 
        0, harrisNumElectron, 1000, particlesElectron
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxElectronB, bulkVyElectronB, bulkVzElectronB, vThElectronB, vThElectronB, vThElectronB, 
        harrisNumElectron, totalNumElectron, 1100, particlesElectron
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxHeavyIonB, bulkVyHeavyIonB, bulkVzHeavyIonB, vThHeavyIonB, vThHeavyIonB, vThHeavyIonB, 
        0, totalNumHeavyIon, 1200, particlesHeavyIon
    );


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), thrust::raw_pointer_cast(B.data())
    );

    hipDeviceSynchronize();
}


int main()
{
    initializeDeviceConstants();

    std::cout << "total number of partices is " << totalNumParticles << std::endl;
    std::cout << std::setprecision(4) 
              << "Omega_ci * t = " << totalStep * dt * omegaCi << std::endl;


    PIC2D pIC2D;

    size_t free_mem = 0;
    size_t total_mem = 0;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

    std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;


    pIC2D.initialize();

    for (int step = 0; step < totalStep+1; step++) {
        if (step % fieldRecordStep == 0) {
            std::cout << std::to_string(step) << " step done : total time is "
                      << std::setprecision(4) << step * dt * omegaCi
                      << " [Omega_ci * t]"
                      << std::endl;
            logfile << std::setprecision(6) << totalTime << std::endl;
            pIC2D.saveFields(
                directoryname, filenameWithoutStep, step
            );
            pIC2D.saveMoments(
                directoryname, filenameWithoutStep, step
            );
        }
        if (isParticleRecord && step % particleRecordStep == 0) {
            pIC2D.saveParticle(
                directoryname, filenameWithoutStep, step
            );
        }
        
        pIC2D.oneStepPeriodicXWallY();

        if (step % 100 == 0) {
            pIC2D.sortParticle();
        }

        totalTime += dt;
    }

    return 0;
}



