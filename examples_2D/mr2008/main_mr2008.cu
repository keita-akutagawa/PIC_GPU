#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <cmath>
#include "../../lib_pic2D_gpu_single/pic2D.hpp"
#include <hip/hip_runtime.h>


std::string directoryname = "results_mr2008";
std::string filenameWithoutStep = "mr2008";
std::ofstream logfile("results_mr2008/log_mr2008.txt");

const int totalStep = 5000;
const int fieldRecordStep = 100;
const int particleRecordStep = 1000;
float totalTime = 0.0f;

const float c = 1.0f;
const float epsilon0 = 1.0f;
const float mu0 = 1.0f;

const int numberDensityIon = 100;
const int numberDensityElectron = 100;

const float B0 = sqrt(static_cast<float>(numberDensityElectron)) / 1.0;

const float mRatio = 9.0f;
const float mElectron = 1.0f;
const float mIon = mRatio * mElectron;

const float tRatio = 1.0f;
const float tElectron = (B0 * B0 / 2.0 / mu0) / (numberDensityIon + numberDensityElectron * tRatio);
const float tIon = tRatio * tElectron;

const float qRatio = -1.0f;
const float qElectron = -1.0f * sqrt(epsilon0 * tElectron / static_cast<float>(numberDensityElectron));
const float qIon = qRatio * qElectron;

const float omegaPe = sqrt(static_cast<float>(numberDensityElectron) * pow(qElectron, 2) / mElectron / epsilon0);
const float omegaPi = sqrt(static_cast<float>(numberDensityIon) * pow(qIon, 2) / mIon / epsilon0);
const float omegaCe = abs(qElectron * B0 / mElectron);
const float omegaCi = qIon * B0 / mIon;

const float debyeLength = sqrt(epsilon0 * tElectron / static_cast<float>(numberDensityElectron) / pow(qElectron, 2));
//追加
const float ionInertialLength = c / omegaPi;

const int nx = int(200.0f * ionInertialLength);
const float dx = 1.0f;
const float xmin = 0.5f * dx; 
const float xmax = nx * dx - 1.0f * dx;

const int ny = int(50.0f * ionInertialLength);
const float dy = 1.0f;
const float ymin = 0.5f * dy; 
const float ymax = ny * dy - 1.0f * dy;

const float dt = 0.5f;

//追加
const float sheatThickness = 1.5f * ionInertialLength;
const float triggerRatio = 0.1f;
const float xPointPosition = 20.0f * ionInertialLength;

//追加
const int harrisNumIon = int(nx * numberDensityIon * 2.0f * sheatThickness);
const int backgroundNumIon = int(0.2f * nx * ny * numberDensityIon);
const int totalNumIon = harrisNumIon + backgroundNumIon;
const int harrisNumElectron = int(nx * numberDensityElectron * 2.0f * sheatThickness);
const int backgroundNumElectron = int(0.2f * nx * ny * numberDensityElectron);
const int totalNumElectron = harrisNumElectron + backgroundNumElectron;
const int totalNumParticles = totalNumIon + totalNumElectron;

const float vThIon = sqrt(2.0f* tIon / mIon);
const float vThElectron = sqrt(2.0f * tElectron / mElectron);
const float bulkVxElectron = 0.0f;
const float bulkVyElectron = 0.0f;
const float bulkVzElectron = c * debyeLength / sheatThickness * sqrt(2.0f / (1.0f + 1.0f/tRatio));
const float bulkVxIon = -bulkVxElectron / tRatio;
const float bulkVyIon = -bulkVyElectron / tRatio;
const float bulkVzIon = -bulkVzElectron / tRatio;

const float vThIonB = sqrt(2.0f * tIon / 10.0f / mIon);
const float vThElectronB = sqrt(2.0f * tElectron / 10.0f / mElectron);
const float bulkVxElectronB = 0.0f;
const float bulkVyElectronB = 0.0f;
const float bulkVzElectronB = 0.0f;
const float bulkVxIonB = 0.0f;
const float bulkVyIonB = 0.0f;
const float bulkVzIonB = 0.0f;


__constant__ int device_totalStep;
__constant__ int device_recordStep;
__device__ float device_totalTime;

__constant__ float device_c;
__constant__ float device_epsilon0;
__constant__ float device_mu0;

__constant__ int device_numberDensityIon;
__constant__ int device_numberDensityElectron;

__constant__ float device_B0;

__constant__ float device_mRatio;
__constant__ float device_mElectron;
__constant__ float device_mIon;

__constant__ float device_tRatio;
__constant__ float device_tElectron;
__constant__ float device_tIon;

__constant__ float device_qRatio;
__constant__ float device_qElectron;
__constant__ float device_qIon;

__constant__ float device_omegaPe;
__constant__ float device_omegaPi;
__constant__ float device_omegaCe;
__constant__ float device_omegaCi;

__constant__ float device_debyeLength;
//追加
__constant__ float device_ionInertialLength;

__constant__ int device_nx;
__constant__ float device_dx;
__constant__ float device_xmin;
__constant__ float device_xmax;

__constant__ int device_ny;
__constant__ float device_dy;
__constant__ float device_ymin;
__constant__ float device_ymax;

__constant__ float device_dt;

//追加
__constant__ float device_sheatThickness;
__constant__ float device_triggerRatio;
__constant__ float device_xPointPosition;

//追加
__constant__ int device_harrisNumIon;
__constant__ int device_backgroundNumIon;
__constant__ int device_totalNumIon;
__constant__ int device_harrisNumElectron;
__constant__ int device_backgroundNumElectron;
__constant__ int device_totalNumElectron;
__constant__ int device_totalNumParticles;

__constant__ float device_vThIon;
__constant__ float device_vThElectron;
__constant__ float device_bulkVxElectron;
__constant__ float device_bulkVyElectron;
__constant__ float device_bulkVzElectron;
__constant__ float device_bulkVxIon;
__constant__ float device_bulkVyIon;
__constant__ float device_bulkVzIon;

__constant__ float device_vThIonB;
__constant__ float device_vThElectronB;
__constant__ float device_bulkVxElectronB;
__constant__ float device_bulkVyElectronB;
__constant__ float device_bulkVzElectronB;
__constant__ float device_bulkVxIonB;
__constant__ float device_bulkVyIonB;
__constant__ float device_bulkVzIonB;


__global__ void initializeField_kernel(
    ElectricField* E, MagneticField* B
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx && j < device_ny) {
        float yCenter = 0.5f * (device_ymax - device_ymin) + device_ymin;
        E[j + device_ny * i].eX = 0.0f;
        E[j + device_ny * i].eY = 0.0f;
        E[j + device_ny * i].eZ = 0.0f;
        B[j + device_ny * i].bX = device_B0 * tanh((j * device_dy - yCenter) / device_sheatThickness)
                                - device_B0 * device_triggerRatio * (j * device_dy - yCenter) / device_sheatThickness
                                * exp(-(pow((i * device_dx - device_xPointPosition), 2) + pow((j * device_dy - yCenter), 2))
                                / pow(2.0f * device_sheatThickness, 2));
        B[j + device_ny * i].bY = device_B0 * device_triggerRatio * (i * device_dx - device_xPointPosition) / device_sheatThickness
                                * exp(-(pow((i * device_dx - device_xPointPosition), 2) + pow((j * device_dy - yCenter), 2))
                                / pow(2.0f * device_sheatThickness, 2)); 
        B[j + device_ny * i].bZ = 0.0f;
    }
}

void PIC2D::initialize()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_ionInertialLength), &ionInertialLength, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_sheatThickness), &sheatThickness, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_triggerRatio), &triggerRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xPointPosition), &xPointPosition, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_harrisNumIon), &harrisNumIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_backgroundNumIon), &backgroundNumIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_harrisNumElectron), &harrisNumElectron, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_backgroundNumElectron), &backgroundNumElectron, sizeof(int));

    initializeParticle.uniformForPositionX(
        0, totalNumIon, 0, particlesIon
    );
    initializeParticle.uniformForPositionX(
        0, totalNumElectron, 100, particlesElectron
    );

    initializeParticle.harrisForPositionY(
        0, harrisNumIon, 200, sheatThickness, particlesIon
    );
    initializeParticle.uniformForPositionY(
        harrisNumIon, totalNumIon, 300, particlesIon
    );
    initializeParticle.harrisForPositionY(
        0, harrisNumElectron, 400, sheatThickness, particlesElectron
    );
    initializeParticle.uniformForPositionY(
        harrisNumElectron, totalNumElectron, 500, particlesElectron
    );

    initializeParticle.maxwellDistributionForVelocity(
        bulkVxIon, bulkVyIon, bulkVzIon, vThIon, vThIon, vThIon, 
        0, harrisNumIon, 600, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxIonB, bulkVyIonB, bulkVzIonB, vThIonB, vThIonB, vThIonB, 
        harrisNumIon, totalNumIon, 700, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxElectron, bulkVyElectron, bulkVzElectron, vThElectron, vThElectron, vThElectron, 
        0, harrisNumElectron, 800, particlesElectron
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxElectronB, bulkVyElectronB, bulkVzElectronB, vThElectronB, vThElectronB, vThElectronB, 
        harrisNumElectron, totalNumElectron, 900, particlesElectron
    );


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), thrust::raw_pointer_cast(B.data())
    );

    hipDeviceSynchronize();
}


int main()
{
    initializeDeviceConstants();

    std::cout << "total number of partices is " << totalNumParticles << std::endl;
    std::cout << std::setprecision(4) 
              << "Omega_ci * t = " << totalStep * dt * omegaCi << std::endl;


    PIC2D pIC2D;

    pIC2D.initialize();


    for (int step = 0; step < totalStep+1; step++) {
        if (step % fieldRecordStep == 0) {
            std::cout << std::to_string(step) << " step done : total time is "
                      << std::setprecision(4) << step * dt * omegaCi
                      << " [Omega_ci * t]"
                      << std::endl;
            logfile << std::setprecision(6) << totalTime << std::endl;
            pIC2D.saveFields(
                directoryname, filenameWithoutStep, step
            );
        }
        if (step % particleRecordStep == 0) {
            pIC2D.saveParticle(
                directoryname, filenameWithoutStep, step
            );
        }
        
        pIC2D.oneStepSymmerticXWallY();

        if (step % 50 == 0) {
            pIC2D.sortParticle();
        }

        totalTime += dt;

        if (step == 0) {
            size_t free_mem = 0;
            size_t total_mem = 0;
            hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

            std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
            std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;
        }
    }

    return 0;
}



