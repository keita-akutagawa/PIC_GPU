#include "const.hpp"


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_c), &c, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_epsilon0), &epsilon0, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mu0), &mu0, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dOfLangdonMarderTypeCorrection), &dOfLangdonMarderTypeCorrection, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dy), &dy, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin), &ymin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax), &ymax, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityIon), &numberDensityIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityElectron), &numberDensityElectron, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityHeavyIon), &numberDensityHeavyIon, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIon), &totalNumIon, sizeof(long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectron), &totalNumElectron, sizeof(long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumHeavyIon), &totalNumHeavyIon, sizeof(long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumParticles), &totalNumParticles, sizeof(long long));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0), &B0, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_mRatio), &mRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mIon), &mIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mElectron), &mElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mHeavyIon), &mHeavyIon, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_tRatio), &tRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tIon), &tIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tElectron), &tElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tHeavyIon), &tHeavyIon, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_qRatio), &qRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qIon), &qIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qElectron), &qElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qHeavyIon), &qHeavyIon, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPe), &omegaPe, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPi), &omegaPi, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCe), &omegaCe, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCi), &omegaCi, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_debyeLength), &debyeLength, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_vThIon), &vThIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThElectron), &vThElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThHeavyIon), &vThHeavyIon, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxIon), &bulkVxIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyIon), &bulkVyIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzIon), &bulkVzIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxElectron), &bulkVxElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyElectron), &bulkVyElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzElectron), &bulkVzElectron, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep), &totalStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime), &totalTime, sizeof(float));
}

