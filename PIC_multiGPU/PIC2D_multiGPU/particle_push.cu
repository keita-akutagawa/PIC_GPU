#include "hip/hip_runtime.h"
#include "particle_push.hpp"


ParticlePush::ParticlePush(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo)
{
}


void ParticlePush::pushVelocity(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    const thrust::device_vector<MagneticField>& B, 
    const thrust::device_vector<ElectricField>& E, 
    float dt
)
{
    pushVelocityOfOneSpecies(
        particlesIon, B, E, qIon, mIon, 
        mPIInfo.existNumIonPerProcs, dt
    );
    pushVelocityOfOneSpecies(
        particlesElectron, B, E, qElectron, mElectron, 
        mPIInfo.existNumElectronPerProcs, dt
    );
}


void ParticlePush::pushPosition(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    float dt
)
{
    pushPositionOfOneSpecies(
        particlesIon, mPIInfo.existNumIonPerProcs, dt
    );
    pushPositionOfOneSpecies(
        particlesElectron, mPIInfo.existNumElectronPerProcs, dt
    );
}


//////////

__device__
ParticleField getParticleFields(
    const MagneticField* B,
    const ElectricField* E, 
    const Particle& particle, 
    const int localNx, const int localNy, const int buffer, 
    const int localSizeX, const int localSizeY, 
    const float xminForProcs, const float xmaxForProcs, 
    const float yminForProcs, const float ymaxForProcs
)
{
    ParticleField particleField;

    float cx1, cx2; 
    int xIndex1, xIndex2;
    float xOverDx;
    float cy1, cy2; 
    int yIndex1, yIndex2;
    float yOverDy;

    xOverDx = (particle.x - xminForProcs + buffer * device_dx) / device_dx;
    yOverDy = (particle.y - yminForProcs + buffer * device_dy) / device_dy;

    xIndex1 = floorf(xOverDx);
    xIndex2 = xIndex1 + 1;
    xIndex2 = (xIndex2 == localSizeX) ? 0 : xIndex2;
    yIndex1 = floorf(yOverDy);
    yIndex2 = yIndex1 + 1;
    yIndex2 = (yIndex2 == localSizeY) ? 0 : yIndex2;

    cx1 = xOverDx - xIndex1;
    cx2 = 1.0f - cx1;
    cy1 = yOverDy - yIndex1;
    cy2 = 1.0f - cy1;

    particleField.bX += B[yIndex1 + localSizeY * xIndex1].bX * cx2 * cy2;
    particleField.bX += B[yIndex2 + localSizeY * xIndex1].bX * cx2 * cy1;
    particleField.bX += B[yIndex1 + localSizeY * xIndex2].bX * cx1 * cy2;
    particleField.bX += B[yIndex2 + localSizeY * xIndex2].bX * cx1 * cy1;

    particleField.bY += B[yIndex1 + localSizeY * xIndex1].bY * cx2 * cy2;
    particleField.bY += B[yIndex2 + localSizeY * xIndex1].bY * cx2 * cy1;
    particleField.bY += B[yIndex1 + localSizeY * xIndex2].bY * cx1 * cy2;
    particleField.bY += B[yIndex2 + localSizeY * xIndex2].bY * cx1 * cy1;

    particleField.bZ += B[yIndex1 + localSizeY * xIndex1].bZ * cx2 * cy2;
    particleField.bZ += B[yIndex2 + localSizeY * xIndex1].bZ * cx2 * cy1;
    particleField.bZ += B[yIndex1 + localSizeY * xIndex2].bZ * cx1 * cy2;
    particleField.bZ += B[yIndex2 + localSizeY * xIndex2].bZ * cx1 * cy1;

    particleField.eX += E[yIndex1 + localSizeY * xIndex1].eX * cx2 * cy2;
    particleField.eX += E[yIndex2 + localSizeY * xIndex1].eX * cx2 * cy1;
    particleField.eX += E[yIndex1 + localSizeY * xIndex2].eX * cx1 * cy2;
    particleField.eX += E[yIndex2 + localSizeY * xIndex2].eX * cx1 * cy1;

    particleField.eY += E[yIndex1 + localSizeY * xIndex1].eY * cx2 * cy2;
    particleField.eY += E[yIndex2 + localSizeY * xIndex1].eY * cx2 * cy1;
    particleField.eY += E[yIndex1 + localSizeY * xIndex2].eY * cx1 * cy2;
    particleField.eY += E[yIndex2 + localSizeY * xIndex2].eY * cx1 * cy1;

    particleField.eZ += E[yIndex1 + localSizeY * xIndex1].eZ * cx2 * cy2;
    particleField.eZ += E[yIndex2 + localSizeY * xIndex1].eZ * cx2 * cy1;
    particleField.eZ += E[yIndex1 + localSizeY * xIndex2].eZ * cx1 * cy2;
    particleField.eZ += E[yIndex2 + localSizeY * xIndex2].eZ * cx1 * cy1;


    return particleField;
}


__global__ void pushVelocityOfOneSpecies_kernel(
    Particle* particlesSpecies, const MagneticField* B, const ElectricField* E, 
    float q, float m, unsigned long long existNumSpecies, float dt, 
    const int localNx, const int localNy, const int buffer, 
    const int localSizeX, const int localSizeY, 
    const float xminForProcs, const float xmaxForProcs, 
    const float yminForProcs, const float ymaxForProcs
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float qOverMTimesDtOver2;
        float tmpForT, tmpForS, tmp1OverC2;
        float vx, vy, vz, gamma;
        float tx, ty, tz;
        float sx, sy, sz;
        float vxMinus, vyMinus, vzMinus;
        float vx0, vy0, vz0;
        float vxPlus, vyPlus, vzPlus; 
        float bx, by, bz;
        float ex, ey, ez;
        ParticleField particleField;

        qOverMTimesDtOver2 = q / m * dt / 2.0f;
        tmp1OverC2 = 1.0f / (device_c * device_c);


        vx = particlesSpecies[i].vx;
        vy = particlesSpecies[i].vy;
        vz = particlesSpecies[i].vz;
        gamma = particlesSpecies[i].gamma;

        particleField = getParticleFields(
            B, E, particlesSpecies[i], 
            localNx, localNy, buffer, 
            localSizeX, localSizeY, 
            xminForProcs, xmaxForProcs, 
            yminForProcs, ymaxForProcs
        );

        bx = particleField.bX;
        by = particleField.bY;
        bz = particleField.bZ; 
        ex = particleField.eX;
        ey = particleField.eY; 
        ez = particleField.eZ;

        tmpForT = qOverMTimesDtOver2 / gamma;
        tx = tmpForT * bx;
        ty = tmpForT * by;
        tz = tmpForT * bz;

        tmpForS = 2.0f / (1.0f + tx * tx + ty * ty + tz * tz);
        sx = tmpForS * tx;
        sy = tmpForS * ty;
        sz = tmpForS * tz;

        vxMinus = vx + qOverMTimesDtOver2 * ex;
        vyMinus = vy + qOverMTimesDtOver2 * ey;
        vzMinus = vz + qOverMTimesDtOver2 * ez;

        vx0 = vxMinus + (vyMinus * tz - vzMinus * ty);
        vy0 = vyMinus + (vzMinus * tx - vxMinus * tz);
        vz0 = vzMinus + (vxMinus * ty - vyMinus * tx);

        vxPlus = vxMinus + (vy0 * sz - vz0 * sy);
        vyPlus = vyMinus + (vz0 * sx - vx0 * sz);
        vzPlus = vzMinus + (vx0 * sy - vy0 * sx);

        vx = vxPlus + qOverMTimesDtOver2 * ex;
        vy = vyPlus + qOverMTimesDtOver2 * ey;
        vz = vzPlus + qOverMTimesDtOver2 * ez;
        gamma = sqrt(1.0f + (vx * vx + vy * vy + vz * vz) * tmp1OverC2);

        particlesSpecies[i].vx = vx;
        particlesSpecies[i].vy = vy;
        particlesSpecies[i].vz = vz;
        particlesSpecies[i].gamma = gamma;
    }
}


void ParticlePush::pushVelocityOfOneSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    const thrust::device_vector<MagneticField>& B,
    const thrust::device_vector<ElectricField>& E, 
    float q, float m, unsigned long long existNumSpecies, 
    float dt
)
{
    float xminForProcs = xmin + (xmax - xmin) / mPIInfo.gridX * mPIInfo.localGridX;
    float xmaxForProcs = xmin + (xmax - xmin) / mPIInfo.gridX * (mPIInfo.localGridX + 1);
    float yminForProcs = ymin + (ymax - ymin) / mPIInfo.gridY * mPIInfo.localGridY;
    float ymaxForProcs = ymin + (ymax - ymin) / mPIInfo.gridY * (mPIInfo.localGridY + 1);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    pushVelocityOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(E.data()), 
        q, m, existNumSpecies, dt, 
        mPIInfo.localNx, mPIInfo.localNy, mPIInfo.buffer, 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        xminForProcs, xmaxForProcs, yminForProcs, ymaxForProcs
    );
    hipDeviceSynchronize();
}


//////////

__global__
void pushPositionOfOneSpecies_kernel(
    Particle* particlesSpecies, const unsigned long long existNumSpecies, 
    const float dt, 
    const float xminForProcs, const float xmaxForProcs, 
    const float yminForProcs, const float ymaxForProcs, 
    const int buffer
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float vx, vy, vz, gamma;
        float xPast, yPast, zPast;
        float x, y, z;
        float dtOverGamma;

        vx = particlesSpecies[i].vx;
        vy = particlesSpecies[i].vy;
        vz = particlesSpecies[i].vz;
        gamma = particlesSpecies[i].gamma;
        xPast = particlesSpecies[i].x;
        yPast = particlesSpecies[i].y;
        zPast = particlesSpecies[i].z;

        dtOverGamma = dt / gamma;
        x = xPast + dtOverGamma * vx;
        y = yPast + dtOverGamma * vy;
        z = zPast + dtOverGamma * vz;

        particlesSpecies[i].x = x;
        particlesSpecies[i].y = y;
        particlesSpecies[i].z = z;

        if (xPast >= xminForProcs + buffer * device_dx && x < xminForProcs + buffer * device_dx) {
            particlesSpecies[i].isMPISendLeftToRight = true;
        }   
        if (xPast <= xmaxForProcs - buffer * device_dx && x > xmaxForProcs - buffer * device_dx) {
            particlesSpecies[i].isMPISendRightToLeft = true;
        }
        if (yPast >= yminForProcs + buffer * device_dy && y < yminForProcs + buffer * device_dy) {
            particlesSpecies[i].isMPISendUpToDown = true;
        }   
        if (yPast <= ymaxForProcs - buffer * device_dy && y > ymaxForProcs - buffer * device_dy) {
            particlesSpecies[i].isMPISendDownToUp = true;
        }
    }
}


void ParticlePush::pushPositionOfOneSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long existNumSpecies, 
    float dt
)
{
    float xminForProcs = xmin + (xmax - xmin) / mPIInfo.gridX * mPIInfo.localGridX;
    float xmaxForProcs = xmin + (xmax - xmin) / mPIInfo.gridX * (mPIInfo.localGridX + 1);
    float yminForProcs = ymin + (ymax - ymin) / mPIInfo.gridY * mPIInfo.localGridY;
    float ymaxForProcs = ymin + (ymax - ymin) / mPIInfo.gridY * (mPIInfo.localGridY + 1);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    pushPositionOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        existNumSpecies, dt, 
        xminForProcs, xmaxForProcs, 
        yminForProcs, ymaxForProcs, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();
}


