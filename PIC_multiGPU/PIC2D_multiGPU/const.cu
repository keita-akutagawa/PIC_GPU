#include "const.hpp"


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_c), &c, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_epsilon0), &epsilon0, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mu0), &mu0, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dOfLangdonMarderTypeCorrection), &dOfLangdonMarderTypeCorrection, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dy), &dy, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin), &ymin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax), &ymax, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityIon), &numberDensityIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityElectron), &numberDensityElectron, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIon), &totalNumIon, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectron), &totalNumElectron, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumParticles), &totalNumParticles, sizeof(unsigned long long));

    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumIonPerProcs), &existNumIonPerProcs, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumElectronPerProcs), &existNumElectronPerProcs, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIonPerProcs), &totalNumIonPerProcs, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectronPerProcs), &totalNumElectronPerProcs, sizeof(unsigned long long));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0), &B0, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_mRatio), &mRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mIon), &mIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mElectron), &mElectron, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_tRatio), &tRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tIon), &tIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tElectron), &tElectron, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_qRatio), &qRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qIon), &qIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qElectron), &qElectron, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPe), &omegaPe, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPi), &omegaPi, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCe), &omegaCe, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCi), &omegaCi, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_debyeLength), &debyeLength, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_vThIon), &vThIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThElectron), &vThElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxIon), &bulkVxIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyIon), &bulkVyIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzIon), &bulkVzIon, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxElectron), &bulkVxElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyElectron), &bulkVyElectron, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzElectron), &bulkVzElectron, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep), &totalStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime), &totalTime, sizeof(float));
}

