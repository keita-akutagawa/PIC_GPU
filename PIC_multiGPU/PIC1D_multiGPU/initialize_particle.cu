#include "hip/hip_runtime.h"
#include "initialize_particle.hpp"
#include <thrust/transform.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <random>


__global__ void uniformForPositionX_kernel(
    Particle* particle, 
    const int nStart, const int nEnd, 
    const double xmin, const double xmax, 
    const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, i, 0, &state);
        double x = hiprand_uniform(&state) * (xmax - xmin) + xmin;
        particle[i + nStart].x = x;
        particle[i + nStart].isExist = true;
    }
}


void InitializeParticle::uniformForPositionX(
    int nStart, 
    int nEnd, 
    double xmin, 
    double xmax, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    uniformForPositionX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        nStart, nEnd, 
        xmin, xmax, 
        seed
    );

    hipDeviceSynchronize();
}



__global__ void maxwellDistributionForVelocity_kernel(
    Particle* particle, 
    const double bulkVxSpecies, const double bulkVySpecies, const double bulkVzSpecies, const double vThSpecies, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz; 
        hiprand_init(seed,           100 * i, 0, &stateVx);
        hiprand_init(seed + 1000000, 100 * i, 0, &stateVy);
        hiprand_init(seed + 2000000, 100 * i, 0, &stateVz);

        double vx, vy, vz;

        while (true) {
            vx = bulkVxSpecies + hiprand_normal(&stateVx) * vThSpecies;
            vy = bulkVySpecies + hiprand_normal(&stateVy) * vThSpecies;
            vz = bulkVzSpecies + hiprand_normal(&stateVz) * vThSpecies;

            if (vx * vx + vy * vy + vz * vz < device_c * device_c) break;
        }

        particle[i + nStart].vx = vx;
        particle[i + nStart].vy = vy;
        particle[i + nStart].vz = vz;
        particle[i + nStart].gamma = sqrt(1.0f + (vx * vx + vy * vy + vz * vz) / (device_c * device_c));
        particle[i + nStart].isExist = true;
    }
}


void InitializeParticle::maxwellDistributionForVelocity(
    double bulkVxSpecies, 
    double bulkVySpecies, 
    double bulkVzSpecies, 
    double vThSpecies, 
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    maxwellDistributionForVelocity_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        bulkVxSpecies, bulkVySpecies, bulkVzSpecies, vThSpecies, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}

