#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include "pic1D.hpp"


PIC1D::PIC1D(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 

      particlesIon     (mPIInfo.totalNumIonPerProcs + 10000), 
      particlesElectron(mPIInfo.totalNumElectronPerProcs + 10000), 
      E         (mPIInfo.localNx + 2 * 1), 
      B         (mPIInfo.localNx + 2 * 1), 
      current   (mPIInfo.localNx + 2 * 1), 
      tmpE      (mPIInfo.localNx + 2 * 1), 
      tmpB      (mPIInfo.localNx + 2 * 1), 
      tmpCurrent(mPIInfo.localNx + 2 * 1), 

      host_particlesIon     (mPIInfo.totalNumIonPerProcs + 10000), 
      host_particlesElectron(mPIInfo.totalNumElectronPerProcs + 10000), 
      host_E      (mPIInfo.localNx + 2 * 1), 
      host_B      (mPIInfo.localNx + 2 * 1), 
      host_current(mPIInfo.localNx + 2 * 1)
{

    hipMalloc(&device_mPIInfo, sizeof(MPIInfo));
    hipMemcpy(device_mPIInfo, &mPIInfo, sizeof(MPIInfo), hipMemcpyHostToDevice);

}


__global__ void getCenterBE_kernel(
    MagneticField* tmpB, ElectricField* tmpE, 
    const MagneticField* B, const ElectricField* E, 
    int localNx
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < localNx) {
        int index = i + 1;

        tmpB[index].bX = B[index].bX;
        tmpB[index].bY = 0.5 * (B[index].bY + B[index - 1].bY);
        tmpB[index].bZ = 0.5 * (B[index].bZ + B[index - 1].bZ);
        tmpE[index].eX = 0.5 * (E[index].eX + E[index - 1].eX);
        tmpE[index].eY = E[index].eY;
        tmpE[index].eZ = E[index].eZ;
    }
}

__global__ void getHalfCurrent_kernel(
    CurrentField* current, const CurrentField* tmpCurrent, 
    int localNx
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < localNx) {
        int index = i + 1;

        current[index].jX = 0.5 * (tmpCurrent[index].jX + tmpCurrent[index + 1].jX);
        current[index].jY = tmpCurrent[index].jY;
        current[index].jZ = tmpCurrent[index].jZ;
    }
}


void PIC1D::oneStep()
{
    
    fieldSolver.timeEvolutionB(B, E, dt / 2.0, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);

    
    sendrecv_field(B, mPIInfo);
    sendrecv_field(E, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((mPIInfo.localNx + threadsPerBlock.x - 1) / threadsPerBlock.x);
    getCenterBE_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpB.data()), 
        thrust::raw_pointer_cast(tmpE.data()), 
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(E.data()), 
        mPIInfo.localNx
    );
    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    sendrecv_field(tmpB, mPIInfo);
    sendrecv_field(tmpE, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);


    particlePush.pushVelocity(
        particlesIon, particlesElectron, tmpB, tmpE, dt, mPIInfo
    );
    
    MPI_Barrier(MPI_COMM_WORLD);
    particlePush.pushPosition(
        particlesIon, particlesElectron, dt / 2.0, mPIInfo
    );
    MPI_Barrier(MPI_COMM_WORLD);
    boundary.periodicBoundaryParticleX(
        particlesIon, particlesElectron, mPIInfo
    );
    MPI_Barrier(MPI_COMM_WORLD);

    currentCalculator.resetCurrent(tmpCurrent);
    currentCalculator.calculateCurrent(
        tmpCurrent, particlesIon, particlesElectron, mPIInfo
    );
    MPI_Barrier(MPI_COMM_WORLD);
    sendrecv_field(tmpCurrent, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);
    getHalfCurrent_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(current.data()), 
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        mPIInfo.localNx
    );
    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_WORLD);
    sendrecv_field(current, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);


    fieldSolver.timeEvolutionB(B, E, dt / 2.0, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);

    fieldSolver.timeEvolutionE(E, B, current, dt, mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);
    
    particlePush.pushPosition(
        particlesIon, particlesElectron, dt / 2.0, mPIInfo
    );
    MPI_Barrier(MPI_COMM_WORLD);
    boundary.periodicBoundaryParticleX(
        particlesIon, particlesElectron, mPIInfo
    );
    MPI_Barrier(MPI_COMM_WORLD);
}


void PIC1D::saveFields(
    std::string directoryname, 
    std::string filenameWithoutStep, 
    int step
)
{

    host_E = E;
    host_B = B;
    host_current = current;
    std::string filenameB, filenameE, filenameCurrent;
    std::string filenameBEnergy, filenameEEnergy;
    double BEnergy = 0.0, EEnergy = 0.0;

    filenameB = directoryname + "/"
             + filenameWithoutStep + "_B_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameE = directoryname + "/"
             + filenameWithoutStep + "_E_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameCurrent = directoryname + "/"
             + filenameWithoutStep + "_current_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameBEnergy = directoryname + "/"
             + filenameWithoutStep + "_BEnergy_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameEEnergy = directoryname + "/"
             + filenameWithoutStep + "_EEnergy_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";


    std::ofstream ofsB(filenameB, std::ios::binary);
    ofsB << std::fixed << std::setprecision(6);
    for (int i = mPIInfo.buffer; i < mPIInfo.localNx + mPIInfo.buffer; i++) {
        ofsB.write(reinterpret_cast<const char*>(&host_B[i].bX), sizeof(double));
        ofsB.write(reinterpret_cast<const char*>(&host_B[i].bY), sizeof(double));
        ofsB.write(reinterpret_cast<const char*>(&host_B[i].bZ), sizeof(double));
        BEnergy += host_B[i].bX * host_B[i].bX + host_B[i].bY * host_B[i].bY + host_B[i].bZ * host_B[i].bZ;
    }
    BEnergy *= 0.5 / mu0;

    std::ofstream ofsE(filenameE, std::ios::binary);
    ofsE << std::fixed << std::setprecision(6);
    for (int i = mPIInfo.buffer; i < mPIInfo.localNx + mPIInfo.buffer; i++) {
        ofsE.write(reinterpret_cast<const char*>(&host_E[i].eX), sizeof(double));
        ofsE.write(reinterpret_cast<const char*>(&host_E[i].eY), sizeof(double));
        ofsE.write(reinterpret_cast<const char*>(&host_E[i].eZ), sizeof(double));
        EEnergy += host_E[i].eX * host_E[i].eX + host_E[i].eY * host_E[i].eY + host_E[i].eZ * host_E[i].eZ;
    }
    EEnergy *= 0.5 * epsilon0;

    std::ofstream ofsCurrent(filenameCurrent, std::ios::binary);
    ofsCurrent << std::fixed << std::setprecision(6);
    for (int i = mPIInfo.buffer; i < mPIInfo.localNx + mPIInfo.buffer; i++) {
        ofsCurrent.write(reinterpret_cast<const char*>(&host_current[i].jX), sizeof(double));
        ofsCurrent.write(reinterpret_cast<const char*>(&host_current[i].jY), sizeof(double));
        ofsCurrent.write(reinterpret_cast<const char*>(&host_current[i].jZ), sizeof(double));
    }

    std::ofstream ofsBEnergy(filenameBEnergy, std::ios::binary);
    ofsBEnergy << std::fixed << std::setprecision(6);
    ofsBEnergy.write(reinterpret_cast<const char*>(&BEnergy), sizeof(double));

    std::ofstream ofsEEnergy(filenameEEnergy, std::ios::binary);
    ofsEEnergy << std::fixed << std::setprecision(6);
    ofsEEnergy.write(reinterpret_cast<const char*>(&EEnergy), sizeof(double));

}


void PIC1D::saveParticle(
    std::string directoryname, 
    std::string filenameWithoutStep, 
    int step
)
{

    host_particlesIon = particlesIon;
    host_particlesElectron = particlesElectron;

    std::string filenameXIon, filenameXElectron;
    std::string filenameVIon, filenameVElectron;
    std::string filenameKineticEnergy;

    filenameXIon = directoryname + "/"
             + filenameWithoutStep + "_x_ion_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameXElectron = directoryname + "/"
             + filenameWithoutStep + "_x_electron_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameVIon = directoryname + "/"
             + filenameWithoutStep + "_v_ion_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameVElectron = directoryname + "/"
             + filenameWithoutStep + "_v_electron_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";
    filenameKineticEnergy = directoryname + "/"
             + filenameWithoutStep + "_KE_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";


    std::ofstream ofsXIon(filenameXIon, std::ios::binary);
    ofsXIon << std::fixed << std::setprecision(6);
    for (int i = 0; i < mPIInfo.existNumIonPerProcs; i++) {
        ofsXIon.write(reinterpret_cast<const char*>(&host_particlesIon[i].x), sizeof(double));
        ofsXIon.write(reinterpret_cast<const char*>(&host_particlesIon[i].y), sizeof(double));
        ofsXIon.write(reinterpret_cast<const char*>(&host_particlesIon[i].z), sizeof(double));
    }

    std::ofstream ofsXElectron(filenameXElectron, std::ios::binary);
    ofsXElectron << std::fixed << std::setprecision(6);
    for (int i = 0; i < mPIInfo.existNumElectronPerProcs; i++) {
        ofsXElectron.write(reinterpret_cast<const char*>(&host_particlesElectron[i].x), sizeof(double));
        ofsXElectron.write(reinterpret_cast<const char*>(&host_particlesElectron[i].y), sizeof(double));
        ofsXElectron.write(reinterpret_cast<const char*>(&host_particlesElectron[i].z), sizeof(double));
    }


    double vx, vy, vz, KineticEnergy = 0.0;

    std::ofstream ofsVIon(filenameVIon, std::ios::binary);
    ofsVIon << std::fixed << std::setprecision(6);
    for (int i = 0; i < mPIInfo.existNumIonPerProcs; i++) {
        vx = host_particlesIon[i].vx;
        vy = host_particlesIon[i].vy;
        vz = host_particlesIon[i].vz;

        ofsVIon.write(reinterpret_cast<const char*>(&vx), sizeof(double));
        ofsVIon.write(reinterpret_cast<const char*>(&vy), sizeof(double));
        ofsVIon.write(reinterpret_cast<const char*>(&vz), sizeof(double));

        KineticEnergy += 0.5 * mIon * (vx * vx + vy * vy + vz * vz);
    }

    std::ofstream ofsVElectron(filenameVElectron, std::ios::binary);
    ofsVElectron << std::fixed << std::setprecision(6);
    for (int i = 0; i < mPIInfo.existNumElectronPerProcs; i++) {
        vx = host_particlesElectron[i].vx;
        vy = host_particlesElectron[i].vy;
        vz = host_particlesElectron[i].vz;

        ofsVElectron.write(reinterpret_cast<const char*>(&vx), sizeof(double));
        ofsVElectron.write(reinterpret_cast<const char*>(&vy), sizeof(double));
        ofsVElectron.write(reinterpret_cast<const char*>(&vz), sizeof(double));
        
        KineticEnergy += 0.5 * mElectron * (vx * vx + vy * vy + vz * vz);
    }

    std::ofstream ofsKineticEnergy(filenameKineticEnergy, std::ios::binary);
    ofsKineticEnergy << std::fixed << std::setprecision(6);
    ofsKineticEnergy.write(reinterpret_cast<const char*>(&KineticEnergy), sizeof(double));

}

