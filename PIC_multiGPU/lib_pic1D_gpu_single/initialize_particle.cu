#include "hip/hip_runtime.h"
#include "initialize_particle.hpp"
#include <thrust/transform.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <random>


__global__ void uniformForPositionX_kernel(
    Particle* particle, 
    const int nStart, const int nEnd, 
    const float xmin, const float xmax, 
    const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState state; 
        hiprand_init(seed, i, 0, &state);
        float x = hiprand_uniform(&state) * (xmax - xmin) + xmin;
        particle[i + nStart].x = x;
        particle[i + nStart].isExist = true;
    }
}


void InitializeParticle::uniformForPositionX(
    int nStart, 
    int nEnd, 
    float xmin, 
    float xmax, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    uniformForPositionX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        nStart, nEnd, 
        xmin, xmax, 
        seed
    );

    hipDeviceSynchronize();
}



__global__ void maxwellDistributionForVelocity_kernel(
    Particle* particle, 
    const float bulkVxSpecies, const float bulkVySpecies, const float bulkVzSpecies, const float vThSpecies, 
    const int nStart, const int nEnd, const int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nEnd - nStart) {
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz; 
        hiprand_init(seed,           100 * i, 0, &stateVx);
        hiprand_init(seed + 1000000, 100 * i, 0, &stateVy);
        hiprand_init(seed + 2000000, 100 * i, 0, &stateVz);

        float vx, vy, vz;

        while (true) {
            vx = bulkVxSpecies + hiprand_normal(&stateVx) * vThSpecies;
            vy = bulkVySpecies + hiprand_normal(&stateVy) * vThSpecies;
            vz = bulkVzSpecies + hiprand_normal(&stateVz) * vThSpecies;

            if (vx * vx + vy * vy + vz * vz < device_c * device_c) break;
        }

        particle[i + nStart].vx = vx;
        particle[i + nStart].vy = vy;
        particle[i + nStart].vz = vz;
        particle[i + nStart].gamma = sqrt(1.0f + (vx * vx + vy * vy + vz * vz) / (device_c * device_c));
        particle[i + nStart].isExist = true;
    }
}


void InitializeParticle::maxwellDistributionForVelocity(
    float bulkVxSpecies, 
    float bulkVySpecies, 
    float bulkVzSpecies, 
    float vThSpecies, 
    int nStart, 
    int nEnd, 
    int seed, 
    thrust::device_vector<Particle>& particlesSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((nEnd - nStart + threadsPerBlock.x - 1) / threadsPerBlock.x);

    maxwellDistributionForVelocity_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        bulkVxSpecies, bulkVySpecies, bulkVzSpecies, vThSpecies, 
        nStart, nEnd, seed
    );

    hipDeviceSynchronize();
}

