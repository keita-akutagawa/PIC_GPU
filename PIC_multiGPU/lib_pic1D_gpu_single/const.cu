#include "const.hpp"


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_c), &c, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_epsilon0), &epsilon0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mu0), &mu0, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityIon), &numberDensityIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityElectron), &numberDensityElectron, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIon), &totalNumIon, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectron), &totalNumElectron, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumParticles), &totalNumParticles, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0), &B0, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_mRatio), &mRatio, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mIon), &mIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mElectron), &mElectron, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_tRatio), &tRatio, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tIon), &tIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tElectron), &tElectron, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_qRatio), &qRatio, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qIon), &qIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qElectron), &qElectron, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPe), &omegaPe, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPi), &omegaPi, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCe), &omegaCe, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCi), &omegaCi, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_debyeLength), &debyeLength, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_vThIon), &vThIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThElectron), &vThElectron, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxIon), &bulkVxIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyIon), &bulkVyIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzIon), &bulkVzIon, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxElectron), &bulkVxElectron, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyElectron), &bulkVyElectron, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzElectron), &bulkVzElectron, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep), &totalStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime), &totalTime, sizeof(double));
}

